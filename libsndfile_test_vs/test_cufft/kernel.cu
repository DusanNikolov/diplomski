#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>

#include <sndfile.hh>

#include <iostream>
#include <iomanip>

using namespace std;

typedef float2 Complex;

hipError_t fftWithCuda();

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

int main()
{

	SndfileHandle* in = new SndfileHandle("D:/Documents/GitHub/Diplomski/libsndfile_test_vs/test_cufft/wav_files/guitar_mono.wav");
	SndfileHandle* out = new SndfileHandle("D:/Documents/GitHub/Diplomski/libsndfile_test_vs/test_cufft/wav_files/out_guit.wav",
		SFM_WRITE, SF_FORMAT_WAV | SF_FORMAT_PCM_16, 1, 44100);
	out->command(SFC_SET_UPDATE_HEADER_AUTO, NULL, SF_TRUE);

	float *in_buf = new float[in->frames()];
	in->readf(in_buf, in->frames());

	long N = in->frames();

	N--;
	N |= N >> 1;
	N |= N >> 2;
	N |= N >> 4;
	N |= N >> 8;
	N |= N >> 16;
	N++;

	long FFT_SIZE = N;
	long ACTUAL_FFT_SIZE = (FFT_SIZE / 2 + 1);

	cout << "FFT_SIZE: " << FFT_SIZE << endl;

	long sig_mem_size = sizeof(hipfftReal)* (ACTUAL_FFT_SIZE * 2);
	long fft_mem_size = sizeof(Complex)* ACTUAL_FFT_SIZE;

	cout << "sig_mem_size: " << sig_mem_size << endl
		<< "fft_mem_size: " << fft_mem_size << endl;

	hipError_t cudaStatus;

	hipfftReal *in_padded;
	cudaStatus = hipMalloc((void**)&in_padded, sig_mem_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 1;
	}

	cudaStatus = hipMemset(in_padded, 0, sig_mem_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		return 2;
	}

	cudaStatus = hipMemcpy(in_padded, in_buf, sizeof(hipfftReal)*100, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 3;
	}

	hipfftHandle dfft, ifft;
	hipfftPlan1d(&dfft, FFT_SIZE, HIPFFT_R2C, 1);
	hipfftPlan1d(&ifft, FFT_SIZE, HIPFFT_C2R, 1);

	
	hipfftExecR2C(dfft, (hipfftReal*)in_padded, (Complex*)in_padded);
	hipfftExecC2R(ifft, (Complex*)in_padded, (hipfftReal*)in_padded);

	float* out_buf = new float[100];

	cudaStatus = hipMemcpy(out_buf, in_padded, sizeof(hipfftReal)*100, hipMemcpyDeviceToHost);


	for (int i = 0; i < 10; i++)
		cout << "in[" << i << "]: " << setprecision(3) << in_buf[i] << '\t' << "out[" << i << "]: " << setprecision(3) << out_buf[i] / FFT_SIZE << endl;


	//out->writef(out_buf, in->frames());


    return 0;
}

hipError_t fftWithCuda() {

	hipError_t cudaStatus = hipSuccess;

	

	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
