#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "device_functions.cuh"


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline hipfftComplex ComplexAdd(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

// Complex scale
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a, float s)
{
	hipfftComplex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, const hipfftComplex *b, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
	}
}

static __global__ void Normalize(hipfftReal *vector, hipfftReal max, int size) {

	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads) {
		vector[i] /= max;
	}

}