#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "device_functions.cuh"


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline hipfftComplex ComplexAdd(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

// Complex scale
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a, float s)
{
	hipfftComplex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, const hipfftComplex *b, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
	}
}


static __global__ void ComplexMultiplyMono(hipfftComplex *out, const hipfftComplex *ir, const hipfftComplex *in, int ir_sz, int in_sz) {

	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < ir_sz; i += numThreads) {
		out[i].x = ir[i].x * in[i % in_sz].x - ir[i].y * in[i % in_sz].y;
		out[i].y = ir[i].x * in[i % in_sz].y + ir[i].y * in[i % in_sz].x;
	}

}

static __global__ void ComplexMultiplyStereo(hipfftComplex *out_l, const hipfftComplex *ir_l, const hipfftComplex *in_l,
	hipfftComplex *out_r, const hipfftComplex *ir_r, const hipfftComplex *in_r, int ir_sz, int in_sz, int trueStereo) {

	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < ir_sz; i += numThreads) {

		//L-L
		out_l[i].x = ir_l[i].x * in_l[i % in_sz].x - ir_l[i].y * in_l[i % in_sz].y;
		out_l[i].y = ir_l[i].x * in_l[i % in_sz].y + ir_l[i].y * in_l[i % in_sz].x;
	
		if (trueStereo == 1) {
			//L-R
			out_l[i].x = ir_r[i].x * in_l[i % in_sz].x - ir_r[i].y * in_l[i % in_sz].y;
			out_l[i].y = ir_r[i].x * in_l[i % in_sz].y + ir_r[i].y * in_l[i % in_sz].x;

			out_l[i].x /= 2;
			out_l[i].y /= 2;

			//R-L
			out_r[i].x = ir_l[i].x * in_r[i % in_sz].x - ir_l[i].y * in_r[i % in_sz].y;
			out_r[i].y = ir_l[i].x * in_r[i % in_sz].y + ir_l[i].y * in_r[i % in_sz].x;

			//R-R
			out_r[i].x = ir_r[i].x * in_r[i % in_sz].x - ir_r[i].y * in_r[i % in_sz].y;
			out_r[i].y = ir_r[i].x * in_r[i % in_sz].y + ir_r[i].y * in_r[i % in_sz].x;

			out_r[i].x /= 2;
			out_r[i].y /= 2;

		}
		else {
			//Quasi stereo
			//R-L
			out_r[i].x = ir_l[i].x * in_r[i % in_sz].x - ir_l[i].y * in_r[i % in_sz].y;
			out_r[i].y = ir_l[i].x * in_r[i % in_sz].y + ir_l[i].y * in_r[i % in_sz].x;

		}
	}

}


//Overlap & Add
static __global__ void OverlapAdd(hipfftReal *dst, int dst_sz, const hipfftReal *src, int src_sz, int M, int N, int odd) {

	//odd == 1 then do odd overlap & add, else do even
	const int numThreads = blockDim.x * gridDim.x;
	//offset, only significant when odd = 1; because odd blocks are shifted to the right by size(offset)
	const int dst_offset = M * odd;
	const int blockID = blockIdx.x * blockDim.x;
	const int threadID = blockID + threadIdx.x;

	const int ind_dst = threadID + dst_offset;
	const int ind_src = (2 * (blockID / N) + odd) * N + threadID - (threadID / N) * N;
	
	
	//dst size  = (IR_blocks - 1) * M + N
	//srr size = IR_blocks * N
	
	//OLA with scaling
	if (threadID < src_sz / 2)
		dst[ind_dst] += src[ind_src] / N;
	
}

//Copy cache
static __global__ void BackupCache(hipfftReal *dst, hipfftReal *src, int count, int size) {

	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadID < size) {
		if (threadID < count) {
			dst[threadID] = src[threadID];
		}
		else {
			dst[threadID] = 0.0f;
		}
	}

}

static __global__ void Normalize(hipfftReal *vector, hipfftReal max, int size) {

	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads) {
		vector[i] /= max;
	}

}



void ComplexMultiplyMono(dim3 gridDim, dim3 blockDim, hipfftComplex *out, const hipfftComplex *ir, const hipfftComplex *in, int ir_sz, int in_sz) {

	ComplexMultiplyMono << <gridDim, blockDim >> >(out, ir, in, ir_sz, in_sz);

}
void ComplexMultiplyStereo(dim3 gridDim, dim3 blockDim, hipfftComplex *out_l, const hipfftComplex *ir_l, const hipfftComplex *in_l,
	hipfftComplex *out_r, const hipfftComplex *ir_r, const hipfftComplex *in_r, int ir_sz, int in_sz, int trueStereo) {

	ComplexMultiplyStereo<<<gridDim, blockDim>>>(out_l, ir_l, in_l, out_r, ir_r, in_r, ir_sz, in_sz, trueStereo);

}

void OverlapAdd(dim3 gridDim, dim3 blockDim, hipfftReal *dst, int dst_sz, const hipfftReal *src, int src_sz, int M, int N, int odd) {

	OverlapAdd<<<gridDim, blockDim>>>(dst, dst_sz, src, src_sz, M, N, odd);

}
void BackupCache(dim3 gridDim, dim3 blockDim, hipfftReal *dst, hipfftReal *src, int count, int size) {

	BackupCache<<<gridDim, blockDim>>>(dst, src, count, size);

}
void Normalize(dim3 gridDim, dim3 blockDim, hipfftReal *vector, hipfftReal max, int size) {

	Normalize<<<gridDim, blockDim>>>(vector, max, size);

}