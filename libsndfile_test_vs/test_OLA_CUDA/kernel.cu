#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "device_functions.cuh"

// Padding functions
int PadData(const Complex *, Complex **, int,
	const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{

	findCudaDevice(argc, (const char **)argv);

	// Allocate host memory for the signal
	Complex *h_signal = (Complex *)malloc(sizeof(Complex)* SIGNAL_SIZE);

	// Initalize the memory for the signal
	for (unsigned int i = 0; i < SIGNAL_SIZE; ++i)
	{
		h_signal[i].x = rand() / (float)RAND_MAX;
		h_signal[i].y = 0;
	}

	// Allocate host memory for the filter
	Complex *h_filter_kernel = (Complex *)malloc(sizeof(Complex)* FILTER_KERNEL_SIZE);

	// Initalize the memory for the filter
	for (unsigned int i = 0; i < FILTER_KERNEL_SIZE; ++i)
	{
		h_filter_kernel[i].x = rand() / (float)RAND_MAX;
		h_filter_kernel[i].y = 0;
	}

	// Pad signal and filter kernel
	Complex *h_padded_signal;
	Complex *h_padded_filter_kernel;
	int new_size = PadData(h_signal, &h_padded_signal, SIGNAL_SIZE,
		h_filter_kernel, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
	int mem_size = sizeof(Complex)* new_size;

	// Allocate device memory for signal
	Complex *d_signal;
	checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));
	// Copy host memory to device
	checkCudaErrors(hipMemcpy(d_signal, h_padded_signal, mem_size,
		hipMemcpyHostToDevice));

	// Allocate device memory for filter kernel
	Complex *d_filter_kernel;
	checkCudaErrors(hipMalloc((void **)&d_filter_kernel, mem_size));

	// Copy host memory to device
	checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
		hipMemcpyHostToDevice));

	// CUFFT plan
	hipfftHandle plan;
	checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_C2C, 1));

	// Transform signal and kernel
	printf("Transforming signal hipfftExecC2C\n");
	checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD));
	checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernel, (hipfftComplex *)d_filter_kernel, HIPFFT_FORWARD));

	// Multiply the coefficients together and normalize the result
	printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
	ComplexPointwiseMulAndScale << <32, 256 >> >(d_signal, d_filter_kernel, new_size, 1.0f / new_size);

	// Check if kernel execution generated and error
	getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

	// Transform signal back
	printf("Transforming signal back hipfftExecC2C\n");
	checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

	// Copy device memory to host
	Complex *h_convolved_signal = h_padded_signal;
	checkCudaErrors(hipMemcpy(h_convolved_signal, d_signal, mem_size,
		hipMemcpyDeviceToHost));

	// Allocate host memory for the convolution result
	Complex *h_convolved_signal_ref = (Complex *)malloc(sizeof(Complex)* SIGNAL_SIZE);

	// Convolve on the host
	Convolve(h_signal, SIGNAL_SIZE,
		h_filter_kernel, FILTER_KERNEL_SIZE,
		h_convolved_signal_ref);

	// check result
	bool bTestResult = sdkCompareL2fe((float *)h_convolved_signal_ref, (float *)h_convolved_signal, 2 * SIGNAL_SIZE, 1e-5f);

	//Destroy CUFFT context
	checkCudaErrors(hipfftDestroy(plan));

	// cleanup memory
	free(h_signal);
	free(h_filter_kernel);
	free(h_padded_signal);
	free(h_padded_filter_kernel);
	free(h_convolved_signal_ref);
	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(d_filter_kernel));

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();
	exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

// Pad data
int PadData(const Complex *signal, Complex **padded_signal, int signal_size,
	const Complex *filter_kernel, Complex **padded_filter_kernel, int filter_kernel_size)
{
	int minRadius = filter_kernel_size / 2;
	int maxRadius = filter_kernel_size - minRadius;
	int new_size = signal_size + maxRadius;

	// Pad signal
	Complex *new_data = (Complex *)malloc(sizeof(Complex)* new_size);
	memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
	memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
	*padded_signal = new_data;

	// Pad filter
	new_data = (Complex *)malloc(sizeof(Complex)* new_size);
	memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
	memset(new_data + maxRadius, 0, (new_size - filter_kernel_size) * sizeof(Complex));
	memcpy(new_data + new_size - minRadius, filter_kernel, minRadius * sizeof(Complex));
	*padded_filter_kernel = new_data;

	return new_size;
}


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
{
	Complex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
	Complex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
	Complex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
	}
}
